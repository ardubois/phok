#include "hip/hip_runtime.h"
#include "erl_nif.h"
#include <cstdlib>
#include <string>
#include <vector>
#include <memory>
#include <cassert>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hiprtc.h>



[[noreturn]] void fail(const std::string& msg, int code) {
    std::cerr << "error: " << msg << " (" << code << ')' << std::endl;
    std::exit(EXIT_FAILURE);
}


char* compile_to_ptx(const char* program_source) {
    hiprtcResult rv;

    // create nvrtc program
    hiprtcProgram prog;
    rv = hiprtcCreateProgram(
        &prog,
        program_source,
        nullptr,
        0,
        nullptr,
        nullptr
    );
    if(rv != HIPRTC_SUCCESS) fail("hiprtcCreateProgram", rv);
    printf("ok\n");
    // compile nvrtc program
    
    //options.push_back("-default-device");
    std::vector<const char*> options = {
        "--include-path=/lib/erlang/usr/include/",
        "--include-path=/usr/include/",
        "--include-path=/usr/lib/",
        "--include-path=/usr/include/x86_64-linux-gnu/",
        "--include-path=/usr/include/c++/11",
        "--include-path=/usr/include/x86_64-linux-gnu/c++/11",
        "--include-path=/usr/include/c++/11/backward",
        "--include-path=/usr/lib/gcc/x86_64-linux-gnu/11/include",
        "--include-path=/usr/include/i386-linux-gnu/",
        "--include-path=/usr/local/include"
 };
    rv = hiprtcCompileProgram(prog, options.size(), options.data());
    if(rv != HIPRTC_SUCCESS) {
        std::size_t log_size;
        rv = hiprtcGetProgramLogSize(prog, &log_size);
        if(rv != HIPRTC_SUCCESS) fail("hiprtcGetProgramLogSize", rv);

        auto log = std::make_unique<char[]>(log_size);
        rv = hiprtcGetProgramLog(prog, log.get());
        if(rv != HIPRTC_SUCCESS) fail("hiprtcGetProgramLog", rv);
        assert(log[log_size - 1] == '\0');

        std::cerr << "Compile error; log:\n" << log.get() << std::endl;

        fail("hiprtcCompileProgram", rv);
    }

    // get ptx code
    std::size_t ptx_size;
    rv = hiprtcGetCodeSize(prog, &ptx_size);
    if(rv != HIPRTC_SUCCESS) fail("hiprtcGetCodeSize", rv);
    char* ptx_source = new char[ptx_size];
    hiprtcGetCode(prog, ptx_source);
  
   
    if(rv != HIPRTC_SUCCESS) fail("hiprtcGetCode", rv);
    assert(ptx_source[ptx_size - 1] == '\0');

    hiprtcDestroyProgram(&prog);

    return ptx_source;
}

const char program_source[] = R"%%%(
//#include <stdint.h>
extern "C" __global__ void f(int* in, int* out) {
    out[threadIdx.x] = in[threadIdx.x];
}
)%%%";

const char program2[] = R"%%%(

__device__
int anon_45cf36d0dd(int x)
{
return ((x + 1));
}


__device__
int cc(int a)
{
return ((a + a));
}


__device__
int g(int a)
{
return (cc((a + a)));
}


__global__
void map_ske(int *a1, int *a2, int size)
{
int id = ((blockIdx.x * blockDim.x) + threadIdx.x);
int r = g(a1[id]);
if((id < size))
{
	//a2[id] = anon_45cf36d0dd(a1[id]);
 // a2[id] = a1[id]+1;
 a2[id] = 1;
}

}


)%%%";

int main() {
    hipError_t err;
    hipDevice_t   device;
    hipCtx_t  context;
    hipModule_t   module;
    hipFunction_t function;
    char       *kernel_name = (char*) "map_ske";

    // initialize CUDA
    err = hipInit(0);
    
    if(err != hipSuccess)  
      { char message[200];
        const char *error;
        hipDrvGetErrorString(err, &error);
        strcpy(message,"Error create_ref_nif: ");
        strcat(message, error);
        printf("%s\n",error);
        exit(-1);
        //enif_raise_exception(env,enif_make_string(env, message, ERL_NIF_LATIN1));
      }

    

    // compile program to ptx
    char* ptx = compile_to_ptx(program2);
   
    printf("%s\n",ptx);

  
  // get device 0

  err = hipDeviceGet(&device, 0); // or some other device on your system
  if (err != hipSuccess) {
        fprintf(stderr, "* Error initializing the CUDA context.\n");
      //  hipCtxDestroy (context);
        exit(-1);
    }

  err = hipCtxCreate(&context, 0, device);
  if (err != hipSuccess) {
        fprintf(stderr, "* Error initializing the CUDA context.\n");
        hipCtxDestroy (context);
        exit(-1);
    }

  // The magic happens here:
  
  err = hipModuleLoadDataEx(&module,  ptx, 0, 0, 0);
  if (err != hipSuccess) {
        fprintf(stderr, "* Error initializing the CUDA context.\n");
        hipCtxDestroy (context);
        exit(-1);
    }

 

  // And here is how you use your compiled PTX
  hipFunction_t kernel_addr;
  err = hipModuleGetFunction(&kernel_addr, module, "_Z7map_skePiS_i");
  if (err != hipSuccess) {
        printf("error: %d\n", err);
        fprintf(stderr, "* Error getting kernel function %s\n", kernel_name);
        hipCtxDestroy (context);
        exit(-1);
    }
  //hipModuleLaunchKernel(kernel_addr, 
   // launch parameters go here
   // kernel arguments go here

   int size = 10;
   int a[size], b[size];
    hipDeviceptr_t d_a, d_b;

   for (int i = 0; i < size; ++i) {
        a[i] = i;
   }     

   err = hipMalloc(&d_a, sizeof(int) * size) ;
    if (err != hipSuccess) {
        printf("error: %d\n", err);
        fprintf(stderr, "* Error getting kernel function %s\n", kernel_name);
        hipCtxDestroy (context);
        exit(-1);
    }

   err = hipMalloc(&d_b, sizeof(int) * size) ;
    if (err != hipSuccess) {
        printf("error: %d\n", err);
        fprintf(stderr, "* Error getting kernel function %s\n", kernel_name);
        hipCtxDestroy (context);
        exit(-1);
    }

   err= hipMemcpyHtoD(d_a, a, sizeof(int) * size) ;
    if (err != hipSuccess) {
        printf("error: %d\n", err);
        fprintf(stderr, "* Error getting kernel function %s\n", kernel_name);
        hipCtxDestroy (context);
        exit(-1);
    }

   void *args[3] = { &d_a, &d_b, &size };

   hipModuleLaunchKernel(function, size, 1, 1,  // Nx1x1 blocks
                                    1, 1, 1,            // 1x1x1 threads
                                    0, 0, args, 0) ;

  hipMemcpyDtoH(b, d_b, sizeof(int) * size) ;

   for (int i = 0; i < size; ++i) {
        printf("result[%d] = %d\n", i, b[i]);
   }     

  hipMemcpyDtoH(a, d_a, sizeof(int) * size) ;

   for (int i = 0; i < size; ++i) {
        printf("result[%d] = %d\n", i, a[i]);
   }      
}  