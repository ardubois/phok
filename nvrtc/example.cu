#include <cstdlib>
#include <string>
#include <vector>
#include <memory>
#include <cassert>
#include <iostream>

#include <hip/hip_runtime.h>

#include <hip/hiprtc.h>

[[noreturn]] void fail(const std::string& msg, int code) {
    std::cerr << "error: " << msg << " (" << code << ')' << std::endl;
    std::exit(EXIT_FAILURE);
}


std::unique_ptr<char[]> compile_to_ptx(const char* program_source) {
    hiprtcResult rv;

    // create nvrtc program
    hiprtcProgram prog;
    rv = hiprtcCreateProgram(
        &prog,
        program_source,
        nullptr,
        0,
        nullptr,
        nullptr
    );
    if(rv != HIPRTC_SUCCESS) fail("nvrtcCreateProgram", rv);
    printf("ok\n");
    // compile nvrtc program
    
    //options.push_back("-default-device");
    std::vector<const char*> options = {
        "--include-path=/lib/erlang/usr/include/",
        "--include-path=/usr/include"
    };
    rv = hiprtcCompileProgram(prog, options.size(), options.data());
    if(rv != HIPRTC_SUCCESS) {
        std::size_t log_size;
        rv = hiprtcGetProgramLogSize(prog, &log_size);
        if(rv != HIPRTC_SUCCESS) fail("nvrtcGetProgramLogSize", rv);

        auto log = std::make_unique<char[]>(log_size);
        rv = hiprtcGetProgramLog(prog, log.get());
        if(rv != HIPRTC_SUCCESS) fail("nvrtcGetProgramLog", rv);
        assert(log[log_size - 1] == '\0');

        std::cerr << "Compile error; log:\n" << log.get() << std::endl;

        fail("nvrtcCompileProgram", rv);
    }

    // get ptx code
    std::size_t ptx_size;
    rv = hiprtcGetCodeSize(prog, &ptx_size);
    if(rv != HIPRTC_SUCCESS) fail("nvrtcGetPTXSize", rv);

    auto ptx = std::make_unique<char[]>(ptx_size);
    rv = hiprtcGetCode(prog, ptx.get());
    if(rv != HIPRTC_SUCCESS) fail("nvrtcGetPTX", rv);
    assert(ptx[ptx_size - 1] == '\0');

    hiprtcDestroyProgram(&prog);

    return ptx;
}

const char program_source[] = R"%%%(
//#include <stdint.h>
extern "C" __global__ void f(int* in, int* out) {
    out[threadIdx.x] = in[threadIdx.x];
}
)%%%";

const char program2[] = R"%%%(
#include "erl_nif.h"


__device__
int anon_45cf36d0dd(int x)
{
return ((x + 1));
}


__device__
int cc(int a)
{
return ((a + a));
}


__device__
int g(int a)
{
return (cc((a + a)));
}


__global__
void map_ske(int *a1, int *a2, int size)
{
int id = ((blockIdx.x * blockDim.x) + threadIdx.x);
int r = g(a1[id]);
if((id < size))
{
	a2[id] = anon_45cf36d0dd(a1[id]);
}

}

extern "C" void map_ske_call(ErlNifEnv *env, const ERL_NIF_TERM argv[], ErlNifResourceType* type,ErlNifResourceType* ftype)
  {

    ERL_NIF_TERM list;
    ERL_NIF_TERM head;
    ERL_NIF_TERM tail;

   // void **fun_res;

    const ERL_NIF_TERM *tuple_blocks;
    const ERL_NIF_TERM *tuple_threads;
    int arity;

    if (!enif_get_tuple(env, argv[1], &arity, &tuple_blocks)) {
      printf ("spawn: blocks argument is not a tuple");
    }

    if (!enif_get_tuple(env, argv[2], &arity, &tuple_threads)) {
      printf ("spawn:threads argument is not a tuple");
    }
    int b1,b2,b3,t1,t2,t3;

    enif_get_int(env,tuple_blocks[0],&b1);
    enif_get_int(env,tuple_blocks[1],&b2);
    enif_get_int(env,tuple_blocks[2],&b3);
    enif_get_int(env,tuple_threads[0],&t1);
    enif_get_int(env,tuple_threads[1],&t2);
    enif_get_int(env,tuple_threads[2],&t3);

    dim3 blocks(b1,b2,b3);
    dim3 threads(t1,t2,t3);

    list= argv[3];

  int **array_res1;
    enif_get_list_cell(env,list,&head,&tail);
    enif_get_resource(env, head, type, (void **) &array_res1);
    int *arg1 = *array_res1;
    list = tail;

    int **array_res2;
    enif_get_list_cell(env,list,&head,&tail);
    enif_get_resource(env, head, type, (void **) &array_res2);
    int *arg2 = *array_res2;
    list = tail;

    enif_get_list_cell(env,list,&head,&tail);
  int arg3;
  enif_get_int(env, head, &arg3);
  list = tail;

   map_ske<<<blocks, threads>>>(arg1,arg2,arg3);
    chipError_terror_gpu = cudaGetLastError();
    if(error_gpu != cudaSuccess)
     { char message[200];
       strcpy(message,"Error kernel call: ");
       strcat(message, cudaGetErrorString(error_gpu));
       enif_raise_exception(env,enif_make_string(env, message, ERL_NIF_LATIN1));
     }
}
)%%%";

int main() {
    hipError_t rv;

    // initialize CUDA
    rv = hipInit(0);
    if(rv != hipSuccess) fail("hipInit", rv);
    printf("inicio\n");
    // compile program to ptx
    auto ptx = compile_to_ptx(program2);
    std::cout << "PTX code:\n" << ptx.get() << std::endl;
}