#include <cstdlib>
#include <string>
#include <vector>
#include <memory>
#include <cassert>
#include <iostream>

#include <hip/hip_runtime.h>

#include <hip/hiprtc.h>

[[noreturn]] void fail(const std::string& msg, int code) {
    std::cerr << "error: " << msg << " (" << code << ')' << std::endl;
    std::exit(EXIT_FAILURE);
}


std::unique_ptr<char[]> compile_to_ptx(const char* program_source) {
    hiprtcResult rv;

    // create nvrtc program
    hiprtcProgram prog;
    rv = hiprtcCreateProgram(
        &prog,
        program_source,
        "program.cu",
        0,
        nullptr,
        nullptr
    );
    if(rv != HIPRTC_SUCCESS) fail("nvrtcCreateProgram", rv);
    printf("ok\n");
    // compile nvrtc program
    
    //options.push_back("-default-device");
    rv = hiprtcCompileProgram(prog, 0, nullptr);
    if(rv != HIPRTC_SUCCESS) {
        std::size_t log_size;
        rv = hiprtcGetProgramLogSize(prog, &log_size);
        if(rv != HIPRTC_SUCCESS) fail("nvrtcGetProgramLogSize", rv);

        auto log = std::make_unique<char[]>(log_size);
        rv = hiprtcGetProgramLog(prog, log.get());
        if(rv != HIPRTC_SUCCESS) fail("nvrtcGetProgramLog", rv);
        assert(log[log_size - 1] == '\0');

        std::cerr << "Compile error; log:\n" << log.get() << std::endl;

        fail("nvrtcCompileProgram", rv);
    }

    // get ptx code
    std::size_t ptx_size;
    rv = hiprtcGetCodeSize(prog, &ptx_size);
    if(rv != HIPRTC_SUCCESS) fail("nvrtcGetPTXSize", rv);

    auto ptx = std::make_unique<char[]>(ptx_size);
    rv = hiprtcGetCode(prog, ptx.get());
    if(rv != HIPRTC_SUCCESS) fail("nvrtcGetPTX", rv);
    assert(ptx[ptx_size - 1] == '\0');

    hiprtcDestroyProgram(&prog);

    return ptx;
}

const char program_source[] = R"%%%(
//#include <stdint.h>
extern "C" __global__ void f(int* in, int* out) {
    out[threadIdx.x] = in[threadIdx.x];
}
)%%%";

int main() {
    hipError_t rv;

    // initialize CUDA
    rv = hipInit(0);
    if(rv != hipSuccess) fail("hipInit", rv);

    // compile program to ptx
    auto ptx = compile_to_ptx(program_source);
    std::cout << "PTX code:\n" << ptx.get() << std::endl;
}